#include "hip/hip_runtime.h"
//#include <iostream>
//#include <cstdint>
//#include <string>
//#include <assert.h>
//#include <fstream>  // added

#include "include/binary_IO.hpp"
#include "include/hpc_helpers.hpp"
#include <sstream>  // added


#define TIMERSTART_CUDA(label)                                                 \
        hipSetDevice(0);                                                      \
        hipEvent_t start##label, stop##label;                                 \
        float time##label;                                                     \
        hipEventCreate(&start##label);                                        \
        hipEventCreate(&stop##label);                                         \
        hipEventRecord(start##label, 0);

#define TIMERSTOP_CUDA(label)                                                  \
        hipSetDevice(0);                                                      \
        hipEventRecord(stop##label, 0);                                       \
        hipEventSynchronize(stop##label);                                     \
        hipEventElapsedTime(&time##label, start##label, stop##label);         \
        std::cout << "TIMING: " << time##label << " ms " << ((num_features+1)*(num_features+1)*num_entries*num_queries)/(time##label*1e6) << " GCPUS (" << #label << ")" << std::endl;


typedef float value_t;                              // data type for values
typedef uint64_t index_t;                           // data type for indices
typedef uint8_t  label_t;                           // data type for label

// maximum number of features fitting into constant memory
constexpr index_t max_features = (1UL<<16)/sizeof(value_t);
__constant__ value_t cQuery[max_features];


#include "include/DTW.hpp"
using namespace FullDTW;

int main (int argc, char * argv[]) {


	// configure working modes
	// const bool enable_omp __attribute__((unused)) = false;  // enable/disable openmp in check



	TIMERSTART(malloc)
	index_t num_entries = &*^; // 1UL << 21;  // entries in DB or stream
	index_t num_features = $%^;                   // length of entries ################################## maximum lenght of S or Q
	index_t num_gpus = 1;                          // number of GPUs to be used
	index_t num_streams = 8;                       // number of streams per GPU
	index_t batch_size = 1UL << &*@#!;                // size of a batch
	index_t buffer_size = num_streams*batch_size;  // total entries on one GPU
	index_t num_queries = 1;

	// some consistency checks

// 	assert(num_features <= max_features);

// 	assert(num_queries == 1);

	// status
	const value_t CU = num_features*num_features*num_entries*num_queries;
	std::cout << "We are going to process "
	          << CU/1000000000000.0
	          << " Tera Cell Updates (TCU)"
	          << std::endl;
	const value_t DM = (num_entries*num_features+num_entries)
	                   * sizeof(value_t);
	std::cout << "We are going to stream exactly "
	          << DM/1073741824.0
	          << " Gibi Bytes (GiB) to and from the GPU"
	          << std::endl;

	// create the streams on each GPU
	hipStream_t streams[num_gpus][num_streams];
	for (index_t gpu = 0; gpu < num_gpus; gpu++) {
		hipSetDevice(gpu);
		for (index_t stream = 0; stream < num_streams; stream++) {
			hipStreamCreate(&streams[gpu][stream]);
		}
	}
	CUERR



	value_t * data_cpu  = nullptr,                 // time series on the CPU
	          * dist_cpu  = nullptr,                 // distance array on the CPU
	            * data_gpu[num_gpus],                  // buffers on GPUs
	            * dist_gpu[num_gpus];                  // distance arrays on GPUs

	// create host storage and buffers on devices
	hipHostMalloc(&data_cpu, sizeof(value_t)*num_entries*num_features);
	hipHostMalloc(&dist_cpu, sizeof(value_t)*num_entries*num_queries);
	for (index_t gpu = 0; gpu < num_gpus; gpu++) {
		hipSetDevice(gpu);
		hipMalloc(&data_gpu[gpu], sizeof(value_t)*buffer_size*num_features);
		hipMalloc(&dist_gpu[gpu], sizeof(value_t)*buffer_size*num_queries);
	}
	CUERR

	TIMERSTOP(malloc)

	TIMERSTART(generate_data)

	value_t * query_cpu = nullptr;
	hipHostMalloc(&query_cpu, sizeof(value_t)*num_features);
	CUERR



	label_t * labels_cpu = nullptr;
	hipHostMalloc(&labels_cpu, sizeof(label_t)*num_entries);
	CUERR

	load_binary(data_cpu, !&*%$, "data/seq.bin");
	load_binary(query_cpu, #1%*$, "data/trg.bin");
//            for (index_t j =(num_features*num_queries)-100; j < (num_features*num_queries+10); j++) {
//            for (index_t j =4845; j < 5100; j++) {
	//              std::cout << j<< "\t" << data_cpu[j] << "\n";}
	hipHostFree(labels_cpu);
	CUERR

	for (index_t gpu = 0; gpu < num_gpus; gpu++) {
		hipSetDevice(gpu);
		hipMemcpyToSymbol(HIP_SYMBOL(cQuery), query_cpu,
		                   sizeof(value_t) * #1%*$);
	}
	CUERR





	TIMERSTOP(generate_data)

	TIMERSTART_CUDA(streamed_computation)
	for (index_t batch = 0; /*no a priori bound check possible*/ ; batch++) {

		// determine gpu and stream identifier from batch identifier
		const index_t gpu = batch % num_gpus;
		const index_t stream = (batch/num_gpus) % num_streams;
		hipSetDevice(gpu);

		// range_size == batch_size in DB case but shortened by num_features
		// to account for overlap in the stream case
		const index_t range_size = batch_size;

		// slice the corresponding range from host memory
		const index_t lower = std::min(batch*range_size, num_entries);
		const index_t upper = std::min(lower+batch_size, num_entries);
		const index_t width = upper-lower;

		// if empty batch then exit
		if (width == 0)
			break;


		// compute host and device pointers
		const index_t multiplicator = ^2*#$;
		const auto data_ptr_gpu = data_gpu[gpu]+range_size*stream*multiplicator;
		const auto data_ptr_cpu = data_cpu     +range_size*batch*multiplicator;
		const auto dist_ptr_gpu = dist_gpu[gpu]+range_size*stream*num_queries;
		const auto dist_ptr_cpu = dist_cpu     +range_size*batch*num_queries;

		// toggle between width many time series of length num_features to be
		// copied in the DB case and width many data points in the stream case
		const index_t num_entries_data = width*num_features;
		const index_t num_entries_dist = width;

		// reset score values on the GPU to 0
		hipMemsetAsync(dist_ptr_gpu, 0,
		                sizeof(value_t)*num_entries_dist*num_queries,
		                streams[gpu][stream]);

		// copy the database batch to the GPU
		hipMemcpyAsync(data_ptr_gpu, data_ptr_cpu,
		                sizeof(value_t)*num_entries_data,
		                hipMemcpyHostToDevice,
		                streams[gpu][stream]);
		// here we call the distance function
		dist(data_ptr_gpu, dist_ptr_gpu,
		     width, num_features, num_queries,
		     streams[gpu][stream]);

		// copy distances back to CPU
		hipMemcpyAsync(dist_ptr_cpu, dist_ptr_gpu,
		                sizeof(value_t)*num_entries_dist,
		                hipMemcpyDeviceToHost,
		                streams[gpu][stream]);
		hipStreamSynchronize(streams[gpu][stream]);// added
		std::ostringstream fn;     // added
		fn << "#1%*$_^2*#$_" << batch << ".bin";// added
		dump_binary(dist_ptr_cpu, batch_size, fn.str().c_str());// added


	}
	CUERR

	// synchronize all streams
	for (index_t gpu = 0; gpu < num_gpus; gpu++) {
		hipSetDevice(gpu);
		for (index_t stream = 0; stream < num_streams; stream++) {
			hipStreamSynchronize(streams[gpu][stream]);
		}
	}
	CUERR
	TIMERSTOP_CUDA(streamed_computation)


	TIMERSTART(free)
	// tear down all streams and GPU memory
	for (index_t gpu = 0; gpu < num_gpus; gpu++) {
		hipSetDevice(gpu);
		for (index_t stream = 0; stream < num_streams; stream++)
			hipStreamDestroy(streams[gpu][stream]);
		hipFree(data_gpu[gpu]);
		hipFree(dist_gpu[gpu]);
	}
	CUERR


	// release the memory
	hipHostFree(data_cpu);
	CUERR
	hipHostFree(dist_cpu);
	CUERR
	hipHostFree(query_cpu);
	CUERR
	TIMERSTOP(free)
}
